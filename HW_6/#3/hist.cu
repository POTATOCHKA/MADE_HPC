
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
using namespace std;

__global__ void calc_hist(int *image_arr, int rows, int cols, int *res)
{

    int pt_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (pt_x < rows && pt_y < cols)
    {
        atomicAdd(res + image_arr[pt_x * cols + pt_y], 1);
    }
}

int *read_txt(int rows, int cols)
{
    int *matr = new int[rows * cols];

    ifstream myfile;
    myfile.open("flopa.txt");

    for (int i = 0; i < rows * cols; ++i)
    {
        myfile >> matr[i];
    }
    return matr;
}

void write(int *arr)
{
    ofstream file;
    file.open("hist.txt");
    for (int i = 0; i < 256; ++i)
    {
        file << arr[i] << ' ';
    }
    file.close();
}

int main()
{
    int width = 606;
    int height = 497;
    int *host_img = read_txt(width, height);
    int *image_on_device;
    int *result_on_host;
    int *result_on_device;
    hipMalloc((void **)&image_on_device, width * height * sizeof(int));
    hipMemcpy(image_on_device, host_img, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&result_on_device, 256 * sizeof(int));

    result_on_host = new int[256];
    for (size_t i = 0; i < 256; i++)
    {
        result_on_host[i] = 0;
    }

    calc_hist<<<dim3(256, 256, 1), dim3(3, 3, 1)>>>(image_on_device, width, height, result_on_device);

    hipMemcpy(result_on_host, result_on_device, 256 * sizeof(int), hipMemcpyDeviceToHost);
    write(result_on_host);
    hipFree(result_on_device);
    hipFree(image_on_device);
    return 0;
}