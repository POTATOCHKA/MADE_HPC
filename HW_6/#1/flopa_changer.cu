
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
using namespace std;

__global__ void filter(int *image_arr, int rows, int cols, int filter_size, int *result)
{

    int pt_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_y = blockIdx.y * blockDim.y + threadIdx.y;
    int sum = filter_size * filter_size;
    if (pt_x < rows - filter_size + 1 && pt_y < cols - filter_size + 1)
    {
        int array_index = pt_x * (cols - filter_size + 1) + pt_y;

        for (int i = -filter_size / 2; i <= filter_size / 2; ++i)
        {
            for (int j = -filter_size / 2; j <= filter_size / 2; ++j)
            {
                int temp_x = (pt_x + (filter_size - 1) / 2 + i);
                int temp_y = pt_y + (filter_size - 1) / 2 + j;
                result[array_index] += image_arr[temp_x * cols + temp_y] / sum;
            }
        }
    }
}

int *read_txt(int rows, int cols)
{
    int *matr = new int[rows * cols];

    ifstream myfile;
    myfile.open("flopa.txt");

    for (int i = 0; i < rows * cols; ++i)
    {
        myfile >> matr[i];
    }
    return matr;
}

void write(int *matr, int rows, int cols)
{
    ofstream file;
    file.open("flopa_out.txt");
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            file << matr[i * cols + j] << ' ';
        }
        file << "\n";
    }
    file.close();
}

int main()
{
    int width = 606;
    int height = 497;
    int *host_img = read_txt(width, height);
    int *image_on_device;
    int *result_on_host;
    int *result_on_device;
    hipMalloc((void **)&image_on_device, width * height * sizeof(int));
    hipMemcpy(image_on_device, host_img, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&result_on_device, (width - 2) * (height - 2) * sizeof(int));

    result_on_host = new int[(width - 2) * (height - 2)];

    filter<<<dim3(256, 256, 1), dim3(3, 3, 1)>>>(image_on_device, width, height, 3, result_on_device);

    hipMemcpy(result_on_host, result_on_device, (width - 2) * (height - 2) * sizeof(int), hipMemcpyDeviceToHost);

    write(result_on_host, width - 2, height - 2);
    hipFree(result_on_device);
    hipFree(image_on_device);
    return 0;
}