
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
    using namespace std;

__global__ void median_filter(int *img, int rows, int cols, int filter_size, int *res)
{

    int pt_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_y = blockIdx.y * blockDim.y + threadIdx.y;
    int double_filter = filter_size * filter_size;
    int temp_arr[1000];
    if (pt_x < rows - filter_size + 1 && pt_y < cols - filter_size + 1)
    {
        int final_arr_idx = pt_x * (cols - filter_size + 1) + pt_y;

        for (int i = -filter_size / 2; i <= filter_size / 2; ++i)
        {
            for (int j = -filter_size / 2; j <= filter_size / 2; ++j)
            {
                int arr_idx = (i + filter_size / 2) * filter_size + j + filter_size / 2;
                int temp_x = pt_x + (filter_size - 1) / 2 + i;
                int temp_y = pt_y + (filter_size - 1) / 2 + j;

                temp_arr[arr_idx] = img[temp_x * cols + temp_y];
            }
        }
        for (int i = 0; i < double_filter - 1; ++i)
            for (int j = 1; j < double_filter; ++j)
                if (temp_arr[i] > temp_arr[j])
                {
                    temp_arr[i] = temp_arr[i] ^ temp_arr[j];
                    temp_arr[j] = temp_arr[j] ^ temp_arr[i];
                    temp_arr[i] = temp_arr[i] ^ temp_arr[j];
                }

        res[final_arr_idx] = temp_arr[double_filter / 2 + 1];
    }
}

int *read_txt(int rows, int cols)
{
    int *matr = new int[rows * cols];

    ifstream myfile;
    myfile.open("flopa.txt");

    for (int i = 0; i < rows * cols; ++i)
    {
        myfile >> matr[i];
    }
    return matr;
}

void write(int *matr, int rows, int cols)
{
    ofstream file;
    file.open("flopa_out.txt");
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            file << matr[i * cols + j] << ' ';
        }
        file << "\n";
    }
    file.close();
}

int main()
{
    int width = 347;
    int height = 320;
    int *host_img = read_txt(width, height);
    int *image_on_device;
    int *result_on_host;
    int *result_on_device;
    hipMalloc((void **)&image_on_device, width * height * sizeof(int));
    hipMemcpy(image_on_device, host_img, width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void **)&result_on_device, (width - 2) * (height - 2) * sizeof(int));

    result_on_host = new int[(width - 2) * (height - 2)];

    median_filter<<<dim3(256, 256, 1), dim3(3, 3, 1)>>>(image_on_device, width, height, 3, result_on_device);

    hipMemcpy(result_on_host, result_on_device, (width - 2) * (height - 2) * sizeof(int), hipMemcpyDeviceToHost);
    write(result_on_host, width - 2, height - 2);
    hipFree(result_on_device);
    hipFree(image_on_device);
    return 0;
}